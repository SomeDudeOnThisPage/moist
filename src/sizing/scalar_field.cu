#include "hip/hip_runtime.h"
#ifndef __SCALAR_FIELD_CUH
#define __SCALAR_FIELD_CUH

#include <cstdint>
#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "scalar_field.hpp"
#include "utils.cuh"

namespace ooc
{
    // https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
    #define cuda_error(ans) { cuda_assert((ans), __FILE__, __LINE__); }
    static inline void cuda_assert(hipError_t code, const char *file, int line, bool abort=true)
    {
       if (code != hipSuccess)
       {
          std::cout << (stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line) << std::endl;
          if (abort) exit(code);
       }
    }

    /**
     * Unoptimized Kernel counting sign changes (0, 1) in a 3d-Array. Each thread counts sign changes in a cube around /radius/ of one pixel.
     * TODO: Set a min-size of some cuboid, so as to not have to store each pixel, but only an average of areas to save memory.
     * TODO: Shared memory 👀. Currently, there's loads of global memory accesses, as each thread currently accesses radius^3 values!
     */
    __global__ static void averageScalarField(const PitchedMatrix3d input, PitchedMatrix3d output, int size_x, int size_y, int size_z, int radius)
    {
        const int local_x = cuda_utils::tidx();
        const int local_y = cuda_utils::tidy();
        const int local_z = cuda_utils::tidz();

        const bool is_debug = local_x == 1 && local_y == 1 && local_z == 0;

        if (is_debug)
        {
            printf("thread %d %d %d\n", local_x, local_y, local_z);
        }

        int sign_changes = 0;
        int visited = 0;
        // TODO: min with max. width/height/depth
        uint16_t last = cuda_utils::pitched::get<uint16_t>(input, max(-radius + local_x, 0), max(-radius + local_y, 0), max(-radius + local_z, 0));

        for (int dz = -radius + local_z; dz <= radius + local_x; dz++)
        {
            for (int dy = -radius + local_y; dy <= radius + local_y; dy++)
            {
                for (int dx = -radius + local_x; dx <= radius + local_z; dx++)
                {
                    if (dx < 0 || dy < 0 || dz < 0)
                    {
                        continue;
                    }

                    if (dx >= size_x || dy >= size_y || dz >= size_z)
                    {
                        continue;
                    }

                    const auto local_data = cuda_utils::pitched::get<uint16_t>(input, dx, dy, dz);
                    if (is_debug)
                    {
                        printf("[1, 1, 0] processing %d %d %d - last sign: %d, local: %d\n", dx, dy, dz, last, local_data);
                    }

                    if (last != local_data)
                    {
                        sign_changes++;
                        last = local_data;
                    }

                    visited++;
                }
            }
        }

        printf("thread %d %d %d counted %d sign changes, visited %d\n", cuda_utils::tidx(), cuda_utils::tidy(), cuda_utils::tidz(), sign_changes, visited);
    }
}

void ooc::generate_scalar_field(std::shared_ptr<TiffData> data)
{
    printf("running...\n");
    PitchedMatrix3d matrix;

    hipExtent extent = make_hipExtent(data->width() * sizeof(uint16_t), data->height(), 1);
    cuda_error(hipMalloc3D(&matrix.ptr, extent));

    uint16_t* host_data = new uint16_t[data->width() * data->height() * 1];
    for (int z = 0; z < 1; z++)
    {
        for (int y = 0; y < data->height(); y++)
        {
            for (int x = 0; x < data->width(); x++)
            {
                int index = z * data->height() * data->width() + y * data->width() + x;
                host_data[index] = data->_data[z][y][x];
            }
        }
    }

    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(host_data, data->width() * sizeof(uint16_t), data->width(), data->height());
    copyParams.dstPtr = matrix.ptr;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyHostToDevice;

    cuda_error(hipMemcpy3D(&copyParams));

    dim3 dim_block(2, 2, 1);
    dim3 dim_grid(1, 1, 1);
    printf("call...\n");
    ooc::averageScalarField<<<dim_grid, dim_block>>>(matrix, /* temp until output*/ matrix, data->width(), data->height(), 1, 2);
    cuda_error(hipPeekAtLastError());
    printf("done...\n");
    cuda_error(hipDeviceSynchronize());
}
#endif // __SCALAR_FIELD_CUH
