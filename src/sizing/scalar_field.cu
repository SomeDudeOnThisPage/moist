#include "hip/hip_runtime.h"
#ifndef __SCALAR_FIELD_CUH
#define __SCALAR_FIELD_CUH

#include <cstdint>
#include <iostream>
#include <stdio.h>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <tiffio.h>

#include "scalar_field.hpp"
#include "utils.cuh"

#define DIV_UP(x, y) (x + y - 1) / y

// #define KERNEL_DEBUG(msg) printf("[%d,%d,%d]: %s\n", cuda_utils::tidx(), cuda_utils::tidy(), cuda_utils::tidz(), msg)

namespace ooc
{
    // https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
    #define cuda_error(ans) { cuda_assert((ans), __FILE__, __LINE__); }
    static inline void cuda_assert(hipError_t code, const char *file, int line, bool abort=true)
    {
       if (code != hipSuccess)
       {
          std::cout << (stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line) << std::endl;
          if (abort) exit(code);
       }
    }

    /**
     * Unoptimized Kernel counting sign changes (0, 1) in a 3d-Array. Each thread counts sign changes in a cube around /radius/ of one pixel.
     * TODO: Set a min-size of some cuboid, so as to not have to store each pixel, but only an average of areas to save memory.
     * TODO: Shared memory 👀. Currently, there's loads of global memory accesses, as each thread currently accesses radius^3 values!
     */
    __global__ static void averageScalarField(uint16_t* i, size_t i_pitch, uint16_t* o, size_t o_pitch, int size_x, int size_y, int size_z, int radius)
    {
        const int local_x = cuda_utils::tidx();
        const int local_y = cuda_utils::tidy();
        const int local_z = cuda_utils::tidz();

        const bool is_debug = local_x == 100 && local_y == 100 && local_z == 0;
        if (local_x >= size_x || local_y >= size_y || local_z >= size_z)
        {
            return;
        }

        if (is_debug)
        {
            printf("thread %d %d %d\n", local_x, local_y, local_z);
        }

        int sign_changes = 0;
        int visited = 0;
        // TODO: min with max. width/height/depth
        uint16_t last = i[max(-radius + local_y, 0) * i_pitch * sizeof(uint16_t) + max(-radius + local_x, 0)];  // cuda_utils::pitched::get<uint16_t>(&input, max(-radius + local_x, 0), max(-radius + local_y, 0), max(-radius + local_z, 0));

        for (int dz = max(0, -radius + local_z); dz < min(size_z, radius + local_z); dz++)
        {
            // TODO: better kernel error macro
            if (is_debug)
            {
                printf("[15, 15, 0] dx_from=%d dx_to=%d dy_from=%d dy_to=%d dz_from=%d dz_to=%d\n",
                    -radius + local_x, radius + local_x, -radius + local_y, radius + local_y, max(0, -radius + local_z), min(size_z, radius + local_z));
            }
            for (int dy = max(0, -radius + local_y); dy < min(size_y, radius + local_y); dy++)
            {
                for (int dx = max(0, -radius + local_x); dx < min(size_x, radius + local_x); dx++)
                {
                    const auto local_data = i[dy * i_pitch * sizeof(uint16_t) + dx]; // cuda_utils::pitched::get<uint16_t>(&input, dx, dy, dz);
                    if (is_debug)
                    {
                        printf("[15, 15, 0] processing %d %d %d - last sign: %d, local: %d\n", dx, dy, dz, last, local_data);
                    }

                    if (last != local_data)
                    {
                        sign_changes++;
                        last = local_data;
                    }

                    visited++;
                }
            }
        }

        if (is_debug)
        {
            printf("thread %d %d %d counted %d sign changes, visited %d\n", cuda_utils::tidx(), cuda_utils::tidy(), cuda_utils::tidz(), sign_changes, visited);
        }

        o[local_y * i_pitch * sizeof(uint16_t) + local_x] = sign_changes * 10000 / radius; //
        // cuda_utils::pitched::set<uint16_t>(&o, o_pitch, local_x, local_y, local_z, sign_changes * 100);
    }
}

void ooc::generate_scalar_field(std::shared_ptr<TiffData> data)
{
    hipExtent extent = make_hipExtent(data->width() * sizeof(uint16_t), data->height(), data->depth());

    uint16_t* host_data = new uint16_t[data->width() * data->height() * 1];
    for (int z = 0; z < data->depth(); z++)
    {
        for (int y = 0; y < data->height(); y++)
        {
            for (int x = 0; x < data->width(); x++)
            {
                int index = z * data->height() * data->width() + y * data->width() + x;
                host_data[index] = data->_data[z][y][x];
            }
        }
    }

    hipPitchedPtr d_input;
    hipPitchedPtr d_output;
    cuda_error(hipMalloc3D(&d_input, extent));
    cuda_error(hipMalloc3D(&d_output, extent));

    hipMemcpy3DParms copy_host_to_device = {0};
    copy_host_to_device.srcPtr = make_hipPitchedPtr(host_data, data->width() * sizeof(uint16_t), data->width(), data->height());
    copy_host_to_device.dstPtr = d_input;
    copy_host_to_device.extent = extent;
    copy_host_to_device.kind = hipMemcpyHostToDevice;
    cuda_error(hipMemcpy3D(&copy_host_to_device));

    dim3 dim_block(std::min((uint32_t) 16, data->width()), std::min((uint32_t) 16, data->height()), std::min((uint32_t) 16, data->depth()));
    dim3 dim_grid(DIV_UP(data->width(), 16), DIV_UP(data->height(), 16), DIV_UP(data->depth(), 16));

    OOC_DEBUG("calling w/ grid dimensions " << DIV_UP(data->width(), 16) << " " << DIV_UP(data->height(), 16) << " " << DIV_UP(data->depth(), 16));
    ooc::averageScalarField<<<dim_grid, dim_block>>>(
        (uint16_t*) d_input.ptr,
        d_input.pitch,
        (uint16_t*) d_output.ptr,
        d_output.pitch,
        data->width(),
        data->height(),
        data->depth(),
        5
    );
    cuda_error(hipPeekAtLastError());
    cuda_error(hipDeviceSynchronize());

    hipMemcpy3DParms copy_device_to_host = {0};
    copy_device_to_host.srcPtr = d_output;
    copy_device_to_host.dstPtr = make_hipPitchedPtr(host_data, data->width() * sizeof(uint16_t), data->width(), data->height());
    copy_device_to_host.extent = extent;
    copy_device_to_host.kind = hipMemcpyDeviceToHost;
    cuda_error(hipMemcpy3D(&copy_device_to_host));

    TIFF* tif = TIFFOpen("test.tif", "w");
    TIFFSetField(tif, TIFFTAG_IMAGEWIDTH, data->width());
    TIFFSetField(tif, TIFFTAG_IMAGELENGTH, data->height());
    TIFFSetField(tif, TIFFTAG_SAMPLESPERPIXEL, 1);
    TIFFSetField(tif, TIFFTAG_BITSPERSAMPLE, 16);
    TIFFSetField(tif, TIFFTAG_ORIENTATION, ORIENTATION_TOPLEFT);
    TIFFSetField(tif, TIFFTAG_PLANARCONFIG, PLANARCONFIG_CONTIG);
    TIFFSetField(tif, TIFFTAG_PHOTOMETRIC, PHOTOMETRIC_MINISBLACK);
    TIFFSetField(tif, TIFFTAG_COMPRESSION, COMPRESSION_NONE);
    TIFFSetField(tif, TIFFTAG_ROWSPERSTRIP, TIFFDefaultStripSize(tif, data->width() * sizeof(uint16_t)));

    std::cout << copy_device_to_host.dstPtr.pitch << " " << data->width() << std::endl;
    for (uint32_t depth = 0; depth < data->depth(); depth++)
    {
        for (uint32_t row = 0; row < data->height(); row++)
        {
            if (TIFFWriteScanline(tif,  &host_data[row * data->width()], row, 0) < 0)
            {
                std::cerr << "Error writing row " << row << "\n";
                TIFFClose(tif);
            }
        }
    }

    TIFFClose(tif);
    free(host_data);
    cuda_error(hipFree(d_input.ptr));
    cuda_error(hipFree(d_output.ptr));
}
#endif // __SCALAR_FIELD_CUH
