#include "hip/hip_runtime.h"
#ifndef __SCALAR_FIELD_CUH
#define __SCALAR_FIELD_CUH

#include <cstdint>
#include <iostream>
#include <stdio.h>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "scalar_field.hpp"
#include "utils.cuh"

#define DIV_UP(x, y) (x + y - 1) / y

// #define KERNEL_DEBUG(msg) printf("[%d,%d,%d]: %s\n", cuda_utils::tidx(), cuda_utils::tidy(), cuda_utils::tidz(), msg)

namespace ooc
{
    // https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
    #define cuda_error(ans) { cuda_assert((ans), __FILE__, __LINE__); }
    static inline void cuda_assert(hipError_t code, const char *file, int line, bool abort=true)
    {
       if (code != hipSuccess)
       {
          std::cout << (stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line) << std::endl;
          if (abort) exit(code);
       }
    }

    /**
     * Unoptimized Kernel counting sign changes (0, 1) in a 3d-Array. Each thread counts sign changes in a cube around /radius/ of one pixel.
     * TODO: Set a min-size of some cuboid, so as to not have to store each pixel, but only an average of areas to save memory.
     * TODO: Shared memory 👀. Currently, there's loads of global memory accesses, as each thread currently accesses radius^3 values!
     */
    __global__ static void averageScalarField(const PitchedMatrix3d input, PitchedMatrix3d output, int size_x, int size_y, int size_z, int radius)
    {
        const int local_x = cuda_utils::tidx();
        const int local_y = cuda_utils::tidy();
        const int local_z = cuda_utils::tidz();

        const bool is_debug = local_x == 100 && local_y == 100 && local_z == 0;
        if (local_x >= size_x || local_y >= size_y || local_z >= size_z)
        {
            return;
        }

        if (is_debug)
        {
            printf("thread %d %d %d\n", local_x, local_y, local_z);
        }

        int sign_changes = 0;
        int visited = 0;
        // TODO: min with max. width/height/depth
        uint16_t last = cuda_utils::pitched::get<uint16_t>(input, max(-radius + local_x, 0), max(-radius + local_y, 0), max(-radius + local_z, 0));

        for (int dz = max(0, -radius + local_z); dz < min(size_z, radius + local_z); dz++)
        {
            // TODO: better kernel error macro
            if (is_debug)
            {
                printf("[15, 15, 0] dx_from=%d dx_to=%d dy_from=%d dy_to=%d dz_from=%d dz_to=%d\n",
                    -radius + local_x, radius + local_x, -radius + local_y, radius + local_y, max(0, -radius + local_z), min(size_z, radius + local_z));
            }
            for (int dy = max(0, -radius + local_y); dy < min(size_y, radius + local_y); dy++)
            {
                for (int dx = max(0, -radius + local_x); dx < min(size_x, radius + local_x); dx++)
                {
                    const auto local_data = cuda_utils::pitched::get<uint16_t>(input, dx, dy, dz);
                    if (is_debug)
                    {
                        printf("[15, 15, 0] processing %d %d %d - last sign: %d, local: %d\n", dx, dy, dz, last, local_data);
                    }

                    if (last != local_data)
                    {
                        sign_changes++;
                        last = local_data;
                    }

                    visited++;
                }
            }
        }

        if (is_debug)
        {
            printf("thread %d %d %d counted %d sign changes, visited %d\n", cuda_utils::tidx(), cuda_utils::tidy(), cuda_utils::tidz(), sign_changes, visited);
        }
    }
}

void ooc::generate_scalar_field(std::shared_ptr<TiffData> data)
{
    PitchedMatrix3d input_matrix;
    PitchedMatrix3d output_matrix;

    hipExtent extent = make_hipExtent(data->width() * sizeof(uint16_t), data->height(), data->depth());
    cuda_error(hipMalloc3D(&input_matrix.ptr, extent));
    cuda_error(hipMalloc3D(&output_matrix.ptr, extent));

    uint16_t* host_data = new uint16_t[data->width() * data->height() * 1];
    for (int z = 0; z < data->depth(); z++)
    {
        for (int y = 0; y < data->height(); y++)
        {
            for (int x = 0; x < data->width(); x++)
            {
                int index = z * data->height() * data->width() + y * data->width() + x;
                host_data[index] = data->_data[z][y][x];
            }
        }
    }

    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(host_data, data->width() * sizeof(uint16_t), data->width(), data->height());
    copyParams.dstPtr = input_matrix.ptr;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyHostToDevice;

    cuda_error(hipMemcpy3D(&copyParams));

    dim3 dim_block(std::min((uint32_t) 16, data->width()), std::min((uint32_t) 16, data->height()), std::min((uint32_t) 16, data->depth()));
    dim3 dim_grid(DIV_UP(data->width(), 16), DIV_UP(data->height(), 16), DIV_UP(data->depth(), 16));
    OOC_DEBUG("calling w/ grid dimensions " << DIV_UP(data->width(), 16) << " " << DIV_UP(data->height(), 16) << " " << DIV_UP(data->depth(), 16));
    ooc::averageScalarField<<<dim_grid, dim_block>>>(input_matrix, /* temp until output */ input_matrix, data->width(), data->height(), data->depth(), 5);
    cuda_error(hipPeekAtLastError());
    cuda_error(hipDeviceSynchronize());
}
#endif // __SCALAR_FIELD_CUH
